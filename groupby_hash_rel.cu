#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/copy.h>
#include <thrust/execution_policy.h>
#include <thrust/device_ptr.h>

#include "cpuGroupby.h"
#include "groupby_hash.cuh"

// is there dynamic size constant memory?
__constant__ reductionType ops_c[512];


#include "groupby_hash_templates.cu"


void groupby_hash_GPU(const int* key_columns_h, int num_key_columns, int num_key_rows,
		      const int* value_columns_h, int num_value_columns, int num_value_rows,
		      reductionType* ops, int num_ops, int* output_keys, int* output_values, int &num_output_rows)
{
  constexpr unsigned int BLOCKDIM = 1024;
  constexpr unsigned int HASH_TABLE_SIZE = 1003;
  constexpr unsigned int GRIDDIM = 40; // 40 as GTX 1080 only have 20 SMs and it can schedule 2048 threads
                                       // change to 56*2 = 112 if testing on Tesla P100
  


  int hashsize_mutiplier = 1;
  hipMallocManaged(&overflow_flag,sizeof(int));
  overflow_flag[0] = 0; // No overflow happens

  // variableAllocating
  int* key_columns_d = NULL;
  int* value_columns_d = NULL;
  int* hash_key_idx_d = NULL;
  int* hash_count_d = NULL;
  int* hash_results_d = NULL;

  gpuErrchk(hipMalloc(&key_columns_d, sizeof(int)*num_key_columns*num_key_rows));
  gpuErrchk(hipMalloc(&value_columns_d, sizeof(int)*num_value_columns*num_value_rows));
  gpuErrchk(hipMalloc(&hash_key_idx_d, sizeof(int)*HASH_TABLE_SIZE*hashsize_mutiplier));
  gpuErrchk(hipMalloc(&hash_count_d, sizeof(int)*HASH_TABLE_SIZE*hashsize_mutiplier));
  gpuErrchk(hipMalloc(&hash_results_d, sizeof(int)*HASH_TABLE_SIZE*hashsize_mutiplier*num_ops));
  
  // initialize values
  gpuErrchk(hipMemcpy(key_columns_d, key_columns_h, sizeof(int)*num_key_columns*num_key_rows, hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(value_columns_d, value_columns_h, sizeof(int)*num_value_columns*num_value_rows, hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(ops_c), ops, sizeof(reductionType) * num_ops));
  initializeVariable<int><<<GRIDDIM, BLOCKDIM>>>(hash_key_idx_d, hash_count_d, hash_results_d, HASH_TABLE_SIZE, num_ops);
  gpuErrchk(hipDeviceSynchronize());

  // fill hash table
  fillTable<int, int><<<GRIDDIM, BLOCKDIM>>>(key_columns_d, num_key_rows, num_key_columns,
					     value_columns_d, num_value_rows, num_value_columns,
					     hash_key_idx_d, hash_count_d, hash_results_d,
					     HASH_TABLE_SIZE, num_ops);

  while(overflow_flag[0] = 1){
    hashsize_mutiplier *= 2;
    hipFree(hash_key_idx_d);
    hipFree(hash_count_d);
    hipFree(hash_results_d);
    gpuErrchk(hipMalloc(&hash_key_idx_d, sizeof(int)*HASH_TABLE_SIZE));
    gpuErrchk(hipMalloc(&hash_count_d, sizeof(int)*HASH_TABLE_SIZE));
    gpuErrchk(hipMalloc(&hash_results_d, sizeof(int)*HASH_TABLE_SIZE*num_ops)); 
  
    gpuErrchk(hipMemcpy(key_columns_d, key_columns_h, sizeof(int)*num_key_columns*num_key_rows, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(value_columns_d, value_columns_h, sizeof(int)*num_value_columns*num_value_rows, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(ops_c), ops, sizeof(reductionType) * num_ops));
    initializeVariable<int><<<GRIDDIM, BLOCKDIM>>>(hash_key_idx_d, hash_count_d, hash_results_d, HASH_TABLE_SIZE, num_ops);
    gpuErrchk(hipDeviceSynchronize());


    fillTable<int, int><<<GRIDDIM, BLOCKDIM>>>(key_columns_d, num_key_rows, num_key_columns,
               value_columns_d, num_value_rows, num_value_columns,
               hash_key_idx_d, hash_count_d, hash_results_d,
               HASH_TABLE_SIZE, num_ops);
    gpuErrchk(hipDeviceSynchronize());

  }
  
  gpuErrchk(hipDeviceSynchronize());

  //shrink the hash table to output array
  //Create array of idices for hash table
  int *seq, *hashTable_idxs;
  hipMalloc((void**)&seq, HASH_TABLE_SIZE*sizeof(int)); //for hash index sequence  
  hipMalloc((void**)&hashTable_idxs, HASH_TABLE_SIZE*sizeof(int));  //for key indexs without -1   
  thrust::device_ptr<int> hash_d_seq = thrust::device_pointer_cast(seq); //for hash index sequence 
  thrust::device_ptr<int> hashTable_idxs_d = thrust::device_pointer_cast(hashTable_idxs); //for key indexs without -1 
  thrust::sequence(thrust::device, hash_d_seq, hash_d_seq + HASH_TABLE_SIZE); //fill hash index seq


  //copy hash idex of keys, removeing -1's which signify not used
//   copy_if(policy, index seq start, index seq end, hash keys for comparison, result containing idx to non -1's, comparator)
  auto newEnd = thrust::copy_if(thrust::device, hash_d_seq, hash_d_seq + HASH_TABLE_SIZE, hash_key_idx_d, hashTable_idxs_d, is_not_neg_1());
  
  num_output_rows = newEnd - hashTable_idxs_d;

  int* output_key_columns_d = NULL;
  hipMalloc(&output_key_columns_d, sizeof(int)*num_key_columns*num_output_rows);
  copyUnique<int><<<GRIDDIM,BLOCKDIM>>>(hashTable_idxs, hash_key_idx_d,key_columns_d, output_key_columns_d, num_output_rows, num_key_columns, num_key_rows);

  int* output_value_columns_d = NULL;
  hipMalloc(&output_value_columns_d, sizeof(int)*num_value_columns*num_output_rows);
  copyValues<int><<<GRIDDIM,BLOCKDIM>>>(hashTable_idxs, hash_results_d,hash_count_d, value_columns_d, output_value_columns_d, num_output_rows, num_value_columns, num_value_rows, num_ops, HASH_TABLE_SIZE);

  gpuErrchk(hipDeviceSynchronize());

  // copy back

  gpuErrchk(hipMemcpy(output_keys,output_key_columns_d,sizeof(int)*num_key_columns*num_output_rows,hipMemcpyDeviceToHost)); 
  gpuErrchk(hipMemcpy(output_values,output_value_columns_d,sizeof(int)*num_value_columns*num_output_rows,hipMemcpyDeviceToHost)); 


  // free elements

  hipFree(key_columns_d);
  hipFree(value_columns_d);
  hipFree(hash_key_idx_d);
  hipFree(hash_count_d);
  hipFree(hash_results_d);
  hipFree(output_key_columns_d);
  hipFree(output_value_columns_d);
  hipFree(seq);
  hipFree(hashTable_idxs);
  
}
